﻿#ifndef _DLL

#include "hip/hip_runtime.h"
#include ""
#include "CUDASortingAlgorithms.cuh"

#include <stdio.h>
#include <vector>
#include <chrono>
#include "SampleArray.h"

template<typename T>
void printList(T* list, size_t len)
{
	std::cout << "{ ";
	for (int i = 0; i < len; i++)
	{
		std::cout << list[i];
		if (i < len - 1) std::cout << ", ";
	}
	std::cout << " }" << std::endl;
}

template<typename T>
using SortingAlgorithmDelegate = void (*)(T*, size_t);

using ElemType = unsigned int;

struct SortAlgorithm
{
	enum class Complexity
	{
		NSquared,
		LessThanNSquared,
		NLogN,
		NLogNSquared,
		Linear,
		TooHigh
	};

	SortAlgorithm(const char* name, SortingAlgorithmDelegate<ElemType> sortAlgorithm, Complexity meanComplexity) : name(name), sortAlgorithm(sortAlgorithm), meanComplexity(meanComplexity) {}

	const char* name;
	SortingAlgorithmDelegate<ElemType> sortAlgorithm;
	Complexity meanComplexity;
};

int main()
{
    CUDASortingAlgorithms::initCuda();
	const size_t arrLen = 29;

	std::vector<ElemType> list = SampleArray::createShuffledArray<ElemType>(arrLen);
	std::vector<ElemType> unsortedList;
	std::vector<ElemType> checkList = SampleArray::createIncrementalArray<ElemType>(list.size());

	std::cout << "Array size : " << list.size() << "\n" << std::endl;

	std::vector<SortAlgorithm> sortAlgorithms({
		SortAlgorithm("Bitonic sort (parallel)", CUDASortingAlgorithms::parallelBitonicSort, SortAlgorithm::Complexity::NLogN),
	});

	for (size_t i = 0; i < sortAlgorithms.size(); i++)
	{
		std::cout << sortAlgorithms[i].name << " : ";

		unsortedList = list;
		auto start = std::chrono::steady_clock::now();
		sortAlgorithms[i].sortAlgorithm(unsortedList.data(), unsortedList.size());
		auto benchmarkTime = std::chrono::steady_clock::now() - start;

		if (benchmarkTime.count() / 1000000 > 1000) std::cout << ((double)benchmarkTime.count() / 1000000000.0) << " s";
		else std::cout << ((double)benchmarkTime.count() / 1000000.0) << " ms";

		if (!CUDASortingAlgorithms::checkSorted(unsortedList.data(), list.size()))
		{
			std::cout << " UNSORTED" << std::endl;
		}
		else if (!CUDASortingAlgorithms::sameArray(unsortedList.data(), checkList.data(), list.size()))
		{
			std::cout << " BROKEN" << std::endl;
		}
		else
		{
			std::cout << std::endl;
		}
		if (list.size() <= 32) printList(unsortedList.data(), unsortedList.size());
	}

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

#endif