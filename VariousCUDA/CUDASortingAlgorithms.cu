#include "hip/hip_runtime.h"
#include "CUDASortingAlgorithms.cuh"

namespace CUDASortingAlgorithms
{
    __declspec(dllexport) bool initCuda()
    {
        hipError_t result = hipSetDevice(0);
        if (result != hipError_t::hipSuccess)
        {
            return false;
        }
        warm_up_gpu << <1, 1024 >> > ();
        return true;
    }

    __declspec(dllexport) void* allocCUDAArray(size_t size)
    {
        void* devArr = nullptr;
        hipMalloc(&devArr, size);
        return devArr;
    }

    __declspec(dllexport) void freeCUDAArray(void* devArr)
    {
        hipFree(devArr);
    }

    void*& getDevArrForSorting()
    {
        return devArrForSorting;
    }

    size_t RoundToNextPowerOf2(size_t val)
    {
        val--;
        val |= (val >> 1);
        val |= (val >> 2);
        val |= (val >> 4);
        val |= (val >> 8);
        val |= (val >> 16);
        val++;
        return val;
    }

    __global__ void warm_up_gpu()
    {
        unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
        float ia, ib;
        ia = ib = 0.0f;
        ib += ia + tid;
    }

	template __declspec(dllexport) void parallelBitonicSort<int>(int* arr, int* end);
	template __declspec(dllexport) void parallelBitonicSort<unsigned int>(unsigned int* arr, unsigned int* end);

    template __declspec(dllexport) void parallelOddEvenMergeSort<int>(int* arr, int* end);
    template __declspec(dllexport) void parallelOddEvenMergeSort<unsigned int>(unsigned int* arr, unsigned int* end);

    template __declspec(dllexport) void parallelPairwiseSortingNetwork<int>(int* arr, int* end);
    template __declspec(dllexport) void parallelPairwiseSortingNetwork<unsigned int>(unsigned int* arr, unsigned int* end);
}